/* ==================================================================
	Programmer: Yicheng Tu (ytu@cse.usf.edu)
	The basic SDH algorithm implementation for 3D data
	To compile: nvcc SDH.c -o SDH in the GAIVI machines
   ==================================================================
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>


#define BOX_SIZE	23000 /* size of the data box on one dimension            */
#define ADDITION_CYCLES 73

/* descriptors for single atom in the tree */
typedef struct atomdesc {
	double x_pos;
	double y_pos;
	double z_pos;
} atom;

// typedef double3 gpu_atom;

typedef struct atomdesc_gpu{ 
	double* x;
	double* y;
	double* z;
} gpu_atom;

typedef struct hist_entry{
	unsigned long long d_cnt;   /* need a long long type as the count might be huge */
} bucket;

typedef struct gpu_hist_entry{
	unsigned long d_cnt;
} gpu_bucket;


bucket * histogram;		/* list of all buckets in the histogram     */
bucket* gpu_histogram;  /* list of all buckets in the GPU histogram */
long long	PDH_acnt;	/* total number of data points              */
int num_buckets;		/* total number of buckets in the histogram */
double   PDH_res;		/* value of w (the width of each bucket)    */
atom * atom_list;		/* list of all data points                  */
gpu_atom gpuAtoms;      /* list of data points in GPU's format      */
double p[32]; 			/* Probability of no collisions             */


/* These are for an old way of tracking time */
struct timezone Idunno;
struct timeval startTime, endTime;


/*
	distance of two points in the atom_list
*/
double p2p_distance(int ind1, int ind2) {

	double x1 = atom_list[ind1].x_pos;
	double x2 = atom_list[ind2].x_pos;
	double y1 = atom_list[ind1].y_pos;
	double y2 = atom_list[ind2].y_pos;
	double z1 = atom_list[ind1].z_pos;
	double z2 = atom_list[ind2].z_pos;

	return sqrt((x1 - x2)*(x1-x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2));
}


/*
	brute-force SDH solution in a single CPU thread
*/
int PDH_baseline() {
	int i, j, h_pos;
	double dist;

	for(i = 0; i < PDH_acnt; i++) {
		for(j = i+1; j < PDH_acnt; j++) {
			dist = p2p_distance(i,j);
			h_pos = (int) (dist / PDH_res);
			histogram[h_pos].d_cnt++;
		}
	}
	return 0;
}


//##############################################################################
// GPU Code
//##############################################################################

__device__ inline double reciprocal_sqrt(double x) {
    double y = rsqrt(x);
    return x * y;
}

__device__ inline double euclidDist(double3 p1, double p2x, double p2y, double p2z)
{
	// Component distances between p1 and p2
	double dx = p1.x - p2x;
	double dy = p1.y - p2y;
	double dz = p1.z - p2z;

	// Straight line distance between points
	return reciprocal_sqrt(dx*dx + dy*dy + dz*dz);
}

/*
	GPU kernel function to compute the PDH for a given set of 3d points
*/
// **TODO** Need to figure out how to do this:
	// Moreover, we vectorize each dimension array by loading multiple floating point coordinate values in one data transmission unit
__global__ void PDH_kernel(gpu_atom dev_atom_list, // Array containing all datapoints
					  bucket* dev_histogram, // Array of bucket counts
					  const int PDH_acnt, // Number of datapoints
					  const int PDH_res, // Bucket size
					  const int num_buckets,
					  const int numHistograms)
{
	extern __shared__ unsigned char sharedMemory[];

	gpu_atom tile;
	tile.x = (double*)sharedMemory;
	tile.y = (double*)sharedMemory + blockDim.x;
	tile.z = (double*)sharedMemory + 2 * blockDim.x;

	int warpOffset = threadIdx.x & 0x1f;
	int histOffset = num_buckets*(warpOffset % numHistograms);
	gpu_bucket* localHist = (gpu_bucket*) ((double*)sharedMemory + 3 * blockDim.x);

	// Initialize local histogram to 0
	for (unsigned i = threadIdx.x; i < num_buckets * numHistograms; i += blockDim.x)
	{
		localHist[i].d_cnt = 0;
	}

	// Check if our current thread index is out of range of the array
	unsigned long long int index = (blockDim.x * blockIdx.x) + threadIdx.x;
	
	// Load this thread's left point into a register
	double3 localPoint;
	if (index < PDH_acnt) {
		localPoint.x = dev_atom_list.x[index];
		localPoint.y = dev_atom_list.y[index];
		localPoint.z = dev_atom_list.z[index];
	}
	else {
		localPoint.x = 0; localPoint.y = 0; localPoint.z = 0;
	}

	for (unsigned long int tileInd = blockIdx.x + 1; tileInd < gridDim.x; tileInd++)
	{
		// Load next tile into shared memory
		unsigned long int tileIndex = (blockDim.x * tileInd) + threadIdx.x;
		if (tileIndex < PDH_acnt)
		{
			tile.x[threadIdx.x] = dev_atom_list.x[tileIndex];
			tile.y[threadIdx.x] = dev_atom_list.y[tileIndex];
			tile.z[threadIdx.x] = dev_atom_list.z[tileIndex];
		}	
		__syncthreads();
		

		// Find distance from thread's point to all points in tile
		for (int i = 0; i < blockDim.x; i++)
		{	
			unsigned long long int ind = (blockDim.x * tileInd) + i;
			if (ind < PDH_acnt) {
				// Straight line distance between points
				double dist = euclidDist(localPoint, tile.x[i], tile.y[i], tile.z[i]);
			
				// Determine which bucket it should go into
				int bucket = (int) (dist / PDH_res);
				
				atomicAdd((unsigned long long *) &(localHist[histOffset + bucket].d_cnt), (unsigned long long) 1);
			}
		}
		__syncthreads();
		
	}

	// Every thread store its assigned point into tile
	tile.x[threadIdx.x] = localPoint.x;
	tile.y[threadIdx.x] = localPoint.y;
	tile.z[threadIdx.x] = localPoint.z;
	__syncthreads();

	// Find intra point distances
	// **TODO** Balance the intra point distance calculation
	// Balancing is causing some histogram blocks to count more. Almost all is clustered in specific ones. 
	// We myst be iterating too many times.
	// for (int i = 1; i <= blockDim.x / 2; i++) 
	// {
	// 	int tileIndex = (threadIdx.x + i) % blockDim.x;
	// 	unsigned long long int ind = (blockDim.x * blockIdx.x) + tileIndex;
	// 	if (ind < PDH_acnt && (i <= blockDim.x / 2 - 1 || threadIdx.x < (blockDim.x / 2)))
	// 	{
	// 		double dist = euclidDist(localPoint, tile.x[i], tile.y[i], tile.z[i]);
	// 		int bucket = (int) (dist / PDH_res);
	// 		atomicAdd((unsigned long long *) &(localHist[histOffset + bucket].d_cnt), (unsigned long long) 1);
	// 	}
	// }

	for (int i = threadIdx.x + 1; i < blockDim.x; i++) 
	{
		unsigned long long int ind = (blockDim.x * blockIdx.x) + i;

		if (ind < PDH_acnt)
		{
			double dist = euclidDist(localPoint, tile.x[i], tile.y[i], tile.z[i]);
			int bucket = (int) (dist / PDH_res);

			atomicAdd((unsigned long long *) &(localHist[histOffset + bucket].d_cnt), (unsigned long long) 1);
		}
	}

	__syncthreads();


	// Merging private histogram copies into a single copy
	for (unsigned int curBucket = 0; curBucket < num_buckets; curBucket++)
	{
		for (unsigned int stride = numHistograms/2; stride > 0; stride /= 2) 
		{
			if (threadIdx.x < stride)
			{
				localHist[curBucket + num_buckets*threadIdx.x].d_cnt += localHist[curBucket + num_buckets*threadIdx.x + stride*num_buckets].d_cnt; 
			}
			__syncthreads();
		}
	}

	// Copy local output to global memory
	for (int i = threadIdx.x; i < num_buckets; i += blockDim.x)
	{
		// **TODO** Use a faster tree based reduction algorithm
		// atomicAdd(&(dev_histogram[i].d_cnt), (unsigned long long) localHist[i].d_cnt);
		dev_histogram[blockIdx.x * num_buckets + i].d_cnt = localHist[i].d_cnt;
	}

	// Parallel reduction

	

	for (unsigned int curBucket = blockIdx.x; curBucket < num_buckets; curBucket += gridDim.x)
	{
		for (unsigned int stride = numHistograms/2; stride > 0; stride /= 2) // Not enough threads for every bucket
		{
			if (threadIdx.x < stride)
			{
				dev_histogram[curBucket + num_buckets*threadIdx.x].d_cnt += dev_histogram[curBucket + num_buckets*threadIdx.x + stride*num_buckets].d_cnt; 
			}
			__syncthreads();
		}
	}
	// unsigned int i = blockIdx.x + num_buckets * threadIdx.x;
	// unsigned int ri = blockIdx.x + 2 * num_buckets * threadIdx.x;  
	// localHist[threadIdx.x].d_cnt = dev_histogram[i].d_cnt + dev_histogram[ri].d_cnt;
	// // Local hist is a gpu_bucket, so will not work.

	// __syncthreads();
	// for (unsigned int stride = gridDim.x / 2; stride > 0; stride /= 2)
	// {
	// 	if (threadIdx.x < stride)
	// 	{
	// 		localHist[threadIdx.x].d_cnt += localHist[threadIdx.x + stride].d_cnt;
	// 	} 
	// 	__syncthreads();
	// }
	// if (threadIdx.x == 0) dev_histogram[blockIdx.x].d_cnt = localHist[0].d_cnt;

}

__global__ void reduction(bucket* dev_histogram, // Array of bucket counts
					  const int num_buckets,
					  const int numHistograms)
{
	// Every Block reduces 1 index of the histogram
	// Offset is then blockIdx.x
	// unsigned int segment = 2*blockDim.x*blockIdx.x;
	// printf("Test\n");
	unsigned int i = blockIdx.x + num_buckets * threadIdx.x;
	unsigned int ri = blockIdx.x + 2 * num_buckets * threadIdx.x;  
	// Just need to figure out the i value, 
	// then rest of reduction should work fine

	__shared__ bucket input_s[64];
	input_s[threadIdx.x].d_cnt = dev_histogram[i].d_cnt + dev_histogram[ri].d_cnt;
	__syncthreads();

	for (unsigned int stride = blockDim.x / 2; stride > 0; stride /= 2)
	{
		if (threadIdx.x < stride)
		{
			input_s[threadIdx.x].d_cnt += input_s[threadIdx.x + stride].d_cnt;
		}
		__syncthreads();
	}	

	//Final sum for that bucket will be in index 0
	if (threadIdx.x == 0) atomicAdd(&dev_histogram[blockIdx.x % num_buckets].d_cnt, (unsigned long long) input_s[0].d_cnt);
}


double findLatency(const int k, const int cl = ADDITION_CYCLES)
{
	if (k == 1) return cl;
	return p[k]*cl + (1.5-p[k])*findLatency(k-1, cl + ADDITION_CYCLES); // **TODO** I don't like this
	// Want to find a way to make collisions more impactful without subtracting from 1.5
}

int findRounds(const unsigned int blockSize, const unsigned long long int numBlocks, const size_t sizeHistogram, int k)
{
	int numHistograms = 32 / k;

	int device;
    hipDeviceProp_t prop;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

	int blocksPerSM;
	hipOccupancyMaxActiveBlocksPerMultiprocessor(&blocksPerSM, PDH_kernel, blockSize, numHistograms * sizeHistogram + 3 * sizeof(double) * blockSize);
	
	//number of threads that can be run in each round in a single multiprocessor 
	unsigned long int occupancy = blocksPerSM * blockSize;
	
	int numThreads = blockSize * numBlocks;
	int numMultiprocessors = prop.multiProcessorCount;

	double denominator = numMultiprocessors * occupancy;
	return (numThreads + denominator - 1 ) / denominator;
}

/*
	Finds the optimal number of histogram copies based on latency and and occupancy
*/
int findNumHistograms(const unsigned int blockSize, const unsigned long long int numBlocks, const size_t sizeHistogram) 
{
	int bestk;
	double minLR = INFINITY;

	for (int k = 1; k <= 32; k++) { // Initialize probabilities
		p[k] = exp(-(k*(k-1))/(double)(2*num_buckets));
		// printf("p[%d] = %lf\n", k, p[k]);
	}

	for (int k = 1; k <= 32; k *= 2)
	{
		// L Value
		double L = findLatency(k);

		// R Value
		int R = findRounds(blockSize, numBlocks, sizeHistogram, k); 

		// Find L x R (Total time)
		double LR = L * R;
		
		// printf("NumHist: %d, L: %lf R: %d, LxR: %lf\n", 32 / k, L, R, LR);

		// Find if LxR is smaller than current min
		if (LR > 0 && LR < minLR)
		{
			minLR = LR;
			bestk = k;
		}
	}

	return 32 / bestk;
}


/*
	Wrapper for the PDH gpu kernel function
	Returns the time taken to run CUDA kernel
*/
float PDH_gpu(const unsigned int blockSize = 64)
{
	const size_t sizeAtomList = sizeof(double)*PDH_acnt;
	const size_t sizeHistogram = sizeof(gpu_bucket)*num_buckets;

	// Allocating Memory
	gpu_atom dev_atom_list;
	hipMalloc((void**) &(dev_atom_list.x), sizeAtomList);
	hipMalloc((void**) &(dev_atom_list.y), sizeAtomList);
	hipMalloc((void**) &(dev_atom_list.z), sizeAtomList);

	// Copying input values to gpu atom list
	hipMemcpy(dev_atom_list.x, gpuAtoms.x, sizeAtomList, hipMemcpyHostToDevice);
	hipMemcpy(dev_atom_list.y, gpuAtoms.y, sizeAtomList, hipMemcpyHostToDevice);
	hipMemcpy(dev_atom_list.z, gpuAtoms.z, sizeAtomList, hipMemcpyHostToDevice);

	// Need 1 thread per point
	const unsigned long long int numBlocks = (PDH_acnt + blockSize - 1) / blockSize;

	// printf("Size of the histogram: %ld\n", sizeHistogram);
	int numHistograms = findNumHistograms(blockSize, numBlocks, sizeHistogram);
	size_t amountSharedMemory = sizeHistogram * numHistograms + 3 * sizeof(double) * blockSize; 
	// printf("Num hist: %d\n", numHistograms);

	bucket* dev_histogram;
	hipMalloc((void**) &dev_histogram, sizeHistogram * numBlocks);
	hipMemset(dev_histogram, 0, sizeHistogram * numBlocks);

	// Start timing
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	PDH_kernel<<<numBlocks, blockSize, amountSharedMemory>>>
		(dev_atom_list, dev_histogram, PDH_acnt, PDH_res, num_buckets, numHistograms);

	// Record end time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	// Calculate total time spent computing
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Copy output histogram from global to cpu mem
	hipMemcpy(gpu_histogram, dev_histogram, sizeHistogram, hipMemcpyDeviceToHost);
	hipFree(dev_atom_list.x);
	hipFree(dev_atom_list.y);
	hipFree(dev_atom_list.z);
	hipFree(dev_histogram);

	return elapsedTime;
}

//##############################################################################


/*
	set a checkpoint and show the (natural) running time in seconds
*/
double report_running_time() {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("\nRunning time for CPU version (in seconds): %ld.%06ld", sec_diff, usec_diff);
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}

double report_gpu_running_time(float elapsedTimeMS) {
	// Convert miliseconds to seconds
	double elapsedTimeS = elapsedTimeMS / 1000.0;
	printf("\nRunning time for GPU version (in seconds): %lf", elapsedTimeS);
	return elapsedTimeS;
}


/*
	print the counts in all buckets of the histogram
*/
void output_histogram(){
	int i;
	long long total_cnt = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", histogram[i].d_cnt);
		total_cnt += histogram[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}

void gpu_output_histogram(){
	int i;
	long long total_cnt = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", gpu_histogram[i].d_cnt);
		total_cnt += gpu_histogram[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}


/*
	Compute and display the difference between the CPU and GPU histograms
*/
void compare_histograms(bucket *cpu_hist, bucket *gpu_hist) {
    bool different = false;
	printf("\nDifference between CPU and GPU histograms:");
    for (int i = 0; i < num_buckets; i++) {
        long long diff = cpu_hist[i].d_cnt - gpu_hist[i].d_cnt;
		if (diff != 0) different = true;
        if (i % 5 == 0)
            printf("\n%02d: ", i);
        printf("%15lld ", diff);
        if (i != num_buckets - 1)
            printf("| ");
    }
    printf("\n");
	(different)? printf("Different\n") : printf("Not different\n");
}


int main(int argc, char **argv)
{
	int i;

	PDH_acnt = atoi(argv[1]);
	PDH_res	 = atof(argv[2]);
	int blockSize = atoi(argv[3]);
// printf("args are %d and %f\n", PDH_acnt, PDH_res);

	num_buckets = (int)(BOX_SIZE * 1.732 / PDH_res) + 1;
	histogram = (bucket *)malloc(sizeof(bucket)*num_buckets);
	gpu_histogram = (bucket *)malloc(sizeof(bucket)*num_buckets);

	atom_list = (atom *)malloc(sizeof(atom)*PDH_acnt);

	gpuAtoms.x = (double*)malloc(sizeof(double)*PDH_acnt);
	gpuAtoms.y = (double*)malloc(sizeof(double)*PDH_acnt);
	gpuAtoms.z = (double*)malloc(sizeof(double)*PDH_acnt);

	srand(1);
	/* generate data following a uniform distribution */
	for(i = 0;  i < PDH_acnt; i++) {
		gpuAtoms.x[i] = atom_list[i].x_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		gpuAtoms.y[i] = atom_list[i].y_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		gpuAtoms.z[i] = atom_list[i].z_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
	}
	/* start counting time */
	gettimeofday(&startTime, &Idunno);

	/* call CPU single thread version to compute the histogram */
	PDH_baseline();

	/* check the total running time */
	report_running_time();

	/* print out the histogram */
	output_histogram();

	/* Computing histograms on GPU */
	float elapsedTime = PDH_gpu(blockSize);

	report_gpu_running_time(elapsedTime);

	gpu_output_histogram();

	/* Compare histograms between cpu and gpu */
	compare_histograms(histogram, gpu_histogram);

	return 0;
}

