#include "hip/hip_runtime.h"
/* ==================================================================
	Programmer: Yicheng Tu (ytu@cse.usf.edu)
	The basic SDH algorithm implementation for 3D data
	To compile: nvcc SDH.c -o SDH in the GAIVI machines
   ==================================================================
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>


#define BOX_SIZE	23000 /* size of the data box on one dimension            */

/* descriptors for single atom in the tree */
typedef struct atomdesc {
	double x_pos;
	double y_pos;
	double z_pos;
} atom;

typedef double3 gpu_atom;

typedef struct hist_entry{
	unsigned long long d_cnt;   /* need a long long type as the count might be huge */
} bucket;


bucket * histogram;		/* list of all buckets in the histogram     */
bucket* gpu_histogram;  /* list of all buckets in the GPU histogram */
long long	PDH_acnt;	/* total number of data points              */
int num_buckets;		/* total number of buckets in the histogram */
double   PDH_res;		/* value of w (the width of each bucket)    */
atom * atom_list;		/* list of all data points                  */
gpu_atom* gpuAtoms;     /* list of data points in GPU's format      */


/* These are for an old way of tracking time */
struct timezone Idunno;
struct timeval startTime, endTime;


/*
	distance of two points in the atom_list
*/
double p2p_distance(int ind1, int ind2) {

	double x1 = atom_list[ind1].x_pos;
	double x2 = atom_list[ind2].x_pos;
	double y1 = atom_list[ind1].y_pos;
	double y2 = atom_list[ind2].y_pos;
	double z1 = atom_list[ind1].z_pos;
	double z2 = atom_list[ind2].z_pos;

	return sqrt((x1 - x2)*(x1-x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2));
}


/*
	brute-force SDH solution in a single CPU thread
*/
int PDH_baseline() {
	int i, j, h_pos;
	double dist;

	for(i = 0; i < PDH_acnt; i++) {
		for(j = i+1; j < PDH_acnt; j++) {
			dist = p2p_distance(i,j);
			h_pos = (int) (dist / PDH_res);
			histogram[h_pos].d_cnt++;
		}
	}
	return 0;
}


//##############################################################################
// GPU Code
//##############################################################################

__device__ inline double euclidDist(double3 p1, double3 p2)
{
	// Component distances between p1 and p2
	double dx = p1.x - p2.x;
	double dy = p1.y - p2.y;
	double dz = p1.z - p2.z;

	// Straight line distance between points
	return sqrt(dx*dx + dy*dy + dz*dz);
}

/*
	GPU kernel function to compute the PDH for a given set of 3d points
*/
// Potential optimizations:
// Store copy of output historgram in each of the warps' shared memory

// Threads are not in sync at all. Need to figure out way to make all perform roughly same number of work
// Number of computations range from n to 1

// Going to want to divide distances into sections and figure out how to iterate through

// Threads divide up input into sections and move down each section

__global__ void PDH_kernel(gpu_atom* dev_atom_list, // Array containing all datapoints
						   bucket* dev_histogram, // Array of bucket counts
						   int PDH_acnt, // Number of datapoints
						   int PDH_res) // Bucket size
{
	__shared__ double3 tile[blockDim.x];

	// **TODO** Output privitization in shared memory

	// Check if our current thread index is out of range of the array
	int index = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (index < PDH_acnt)
	{
		// Load this thread's left point into a register
		double3 localPoint = dev_atom_list[index];

		for (int i = blockIdx.x + 1; i < gridDim.x; i++)
		{
			// Load next tile into shared memory
			tile[threadIdx.x] = dev_atom_list[index];
			__syncthreads();

			for (int j = 0; j < blockDim.x; j++)
			{
				// Straight line distance between points
				double dist = euclidDist(localPoint, tile[j]);
				
				// Determine which bucket it should go into
				int bucket = (int) (dist / PDH_res);

				atomicAdd(&(dev_histogram[bucket].d_cnt), (unsigned long long) 1);
			}
		}

		// Find intra point distances
		tile[threadIdx.x] = localPoint;
		__syncthreads();

		// **TODO** Balance the intra point distance calculation
		for (i = threadIdx.x + 1; i < blockDim.x; i++) 
		{
			double dist = euclidDist(localPoint, tile[i]);
			int bucket = (int) (dist / PDH_res);

			atomicAdd(&(dev_histogram[bucket].d_cnt), (unsigned long long) 1);
		}
		__syncthreads();
	}
}


/*
	Wrapper for the PDH gpu kernel function
	Returns the time taken to run CUDA kernel
*/
float PDH_gpu(const int blockSize = 256)
{
	const size_t sizeAtomList = sizeof(gpu_atom)*PDH_acnt;
	const size_t sizeHistogram = sizeof(bucket)*num_buckets;

	// Allocating Memory
	gpu_atom* dev_atom_list;
	hipMalloc((void**) &(dev_atom_list), sizeAtomList);

	// Copying input values to gpu atom list
	hipMemcpy(dev_atom_list, gpuAtoms, sizeAtomList, hipMemcpyHostToDevice);

	bucket* dev_histogram;
	hipMalloc((void**) &dev_histogram, sizeHistogram);
	hipMemset(dev_histogram, 0, sizeHistogram);

	// Need 1 thread per point
	const int numBlocks = (PDH_acnt + blockSize - 1) / blockSize;

	// Start timing
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// Call kernel function (Passing in the array of data)
	PDH_kernel<<<numBlocks, blockSize>>>(dev_atom_list, dev_histogram, PDH_acnt, PDH_res);

	// Record end time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	// Calculate total time spent computing
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);


	// Copy output histogram from global to cpu mem
	hipMemcpy(gpu_histogram, dev_histogram, sizeHistogram, hipMemcpyDeviceToHost);

	hipFree(dev_atom_list);
	hipFree(dev_histogram);

	return elapsedTime;
}

//##############################################################################


/*
	set a checkpoint and show the (natural) running time in seconds
*/
double report_running_time() {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("\nRunning time for CPU version (in seconds): %ld.%06ld", sec_diff, usec_diff);
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}

double report_gpu_running_time(float elapsedTimeMS) {
	// Convert miliseconds to seconds
	double elapsedTimeS = elapsedTimeMS / 1000.0;
	printf("\nRunning time for GPU version (in seconds): %lf", elapsedTimeS);
	return elapsedTimeS;
}


/*
	print the counts in all buckets of the histogram
*/
void output_histogram(){
	int i;
	long long total_cnt = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", histogram[i].d_cnt);
		total_cnt += histogram[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}

void gpu_output_histogram(){
	int i;
	long long total_cnt = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", gpu_histogram[i].d_cnt);
		total_cnt += gpu_histogram[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}


/*
	Compute and display the difference between the CPU and GPU histograms
*/
void compare_histograms(bucket *cpu_hist, bucket *gpu_hist) {
    printf("\nDifference between CPU and GPU histograms:");
    for (int i = 0; i < num_buckets; i++) {
        long long diff = cpu_hist[i].d_cnt - gpu_hist[i].d_cnt;
        if (i % 5 == 0)
            printf("\n%02d: ", i);
        printf("%15lld ", diff);
        if (i != num_buckets - 1)
            printf("| ");
    }
    printf("\n");
}


int main(int argc, char **argv)
{
	int i;

	PDH_acnt = atoi(argv[1]);
	PDH_res	 = atof(argv[2]);
	// int blockSize = atoi(argv[3]);
// printf("args are %d and %f\n", PDH_acnt, PDH_res);

	num_buckets = (int)(BOX_SIZE * 1.732 / PDH_res) + 1;
	histogram = (bucket *)malloc(sizeof(bucket)*num_buckets);
	gpu_histogram = (bucket *)malloc(sizeof(bucket)*num_buckets);

	atom_list = (atom *)malloc(sizeof(atom)*PDH_acnt);

	gpuAtoms = (gpu_atom*)malloc(sizeof(gpu_atom)*PDH_acnt);

	srand(1);
	/* generate data following a uniform distribution */
	for(i = 0;  i < PDH_acnt; i++) {
		atom_list[i].x_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].y_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].z_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;

		gpuAtoms[i].x = atom_list[i].x_pos;
		gpuAtoms[i].y = atom_list[i].y_pos;
		gpuAtoms[i].z = atom_list[i].z_pos;
	}
	/* start counting time */
	gettimeofday(&startTime, &Idunno);

	/* call CPU single thread version to compute the histogram */
	PDH_baseline();

	/* check the total running time */
	report_running_time();

	/* print out the histogram */
	output_histogram();

	/* Computing histograms on GPU */
	float elapsedTime = PDH_gpu();

	report_gpu_running_time(elapsedTime);

	gpu_output_histogram();

	/* Compare histograms between cpu and gpu */
	compare_histograms(histogram, gpu_histogram);

	return 0;
}

